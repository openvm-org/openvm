#include "hip/hip_runtime.h"
#include "adapters/branch.cuh" // Rv32BranchAdapterCols, Rv32BranchAdapterRecord, Rv32BranchAdapter
#include "constants.h"         // RV32_REGISTER_NUM_LIMBS, RV32_CELL_BITS
#include "cores/branch_less_than.cuh"
#include "histogram.cuh"
#include "launcher.cuh"
#include "trace_access.h"

using namespace riscv;

// Concrete type aliases for 32-bit
using Rv32BranchLessThanCoreRecord = BranchLessThanCoreRecord<RV32_REGISTER_NUM_LIMBS>;
using Rv32BranchLessThanCore = BranchLessThanCore<RV32_REGISTER_NUM_LIMBS>;
template <typename T>
using Rv32BranchLessThanCoreCols = BranchLessThanCoreCols<T, RV32_REGISTER_NUM_LIMBS>;

template <typename T> struct BranchLessThanCols {
    Rv32BranchAdapterCols<T> adapter;
    Rv32BranchLessThanCoreCols<T> core;
};

struct BranchLessThanRecord {
    Rv32BranchAdapterRecord adapter;
    Rv32BranchLessThanCoreRecord core;
};

__global__ void blt_tracegen(
    Fp *trace,
    size_t height,
    uint8_t *records,
    size_t num_records,
    uint32_t *rc_ptr,
    uint32_t rc_bins,
    uint32_t *bw_ptr,
    uint32_t bw_bits,
    uint32_t timestamp_max_bits
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    RowSlice row(trace + idx, height);

    if (idx < num_records) {
        auto full = reinterpret_cast<BranchLessThanRecord *>(records)[idx];

        Rv32BranchAdapter adapter(VariableRangeChecker(rc_ptr, rc_bins), timestamp_max_bits);
        adapter.fill_trace_row(row, full.adapter);

        Rv32BranchLessThanCore core(BitwiseOperationLookup(bw_ptr, bw_bits));
        core.fill_trace_row(row.slice_from(COL_INDEX(BranchLessThanCols, core)), full.core);
    } else {
        row.fill_zero(0, sizeof(BranchLessThanCols<uint8_t>));
    }
}

extern "C" int _blt_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    uint8_t *d_records,
    size_t record_len,
    uint32_t *d_rc,
    uint32_t rc_bins,
    uint32_t *d_bw,
    uint32_t bw_bits,
    uint32_t timestamp_max_bits
) {
    assert((height & (height - 1)) == 0);
    assert(height * sizeof(BranchLessThanRecord) >= record_len);
    assert(width == sizeof(BranchLessThanCols<uint8_t>));

    auto [grid, block] = kernel_launch_params(height);
    blt_tracegen<<<grid, block>>>(
        d_trace,
        height,
        d_records,
        record_len / sizeof(BranchLessThanRecord),
        d_rc,
        rc_bins,
        d_bw,
        bw_bits,
        timestamp_max_bits
    );
    return hipGetLastError();
}
