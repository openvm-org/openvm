#include "hip/hip_runtime.h"
#include "launcher.cuh"
#include "poseidon2-air/params.cuh"
#include "poseidon2-air/tracegen.cuh"
#include "primitives/fp_array.cuh"
#include "primitives/trace_access.h"
#include "primitives/utils.cuh"
#include <cstdint>
#include <hipcub/hipcub.hpp>

template <size_t WIDTH, typename PoseidonParams>
__global__ void cukernel_system_poseidon2_tracegen(
    Fp *d_trace,
    size_t trace_height,
    size_t trace_width,
    Fp *d_records,
    uint32_t *d_counts,
    size_t num_records
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    using Poseidon2Row = poseidon2::Poseidon2Row<
        WIDTH,
        PoseidonParams::SBOX_DEGREE,
        PoseidonParams::SBOX_REGS,
        PoseidonParams::HALF_FULL_ROUNDS,
        PoseidonParams::PARTIAL_ROUNDS>;
#ifdef DEBUG
    assert(Poseidon2Row::get_total_size() + 1 == trace_width);
#endif
    if (idx < trace_height) {
        Poseidon2Row row(d_trace + idx, trace_height);
        if (idx < num_records) {
            RowSlice state(d_records + idx * WIDTH, 1);
            poseidon2::generate_trace_row_for_perm(row, state);

            d_trace[idx + Poseidon2Row::get_total_size() * trace_height] = d_counts[idx];
        } else {
            Fp dummy[Poseidon2Row::get_total_size()] = {0};
            RowSlice dummy_row(dummy, 1);
            poseidon2::generate_trace_row_for_perm(row, dummy_row);

            d_trace[idx + Poseidon2Row::get_total_size() * trace_height] = 0;
        }
    }
}

extern "C" int _system_poseidon2_tracegen(
    Fp *d_trace,
    size_t height,
    size_t width,
    Fp *d_records,
    uint32_t *d_counts,
    size_t num_records,
    size_t sbox_regs
) {
    auto [grid, block] = kernel_launch_params(height);

    switch (sbox_regs) {
    case 1:
        cukernel_system_poseidon2_tracegen<16, Poseidon2ParamsS1>
            <<<grid, block, 0, hipStreamPerThread>>>(
                d_trace, height, width, d_records, d_counts, num_records
            );
        break;
    case 0:
        cukernel_system_poseidon2_tracegen<16, Poseidon2ParamsS0>
            <<<grid, block, 0, hipStreamPerThread>>>(
                d_trace, height, width, d_records, d_counts, num_records
            );
        break;
    default:
        return hipErrorInvalidConfiguration;
    }

    return hipGetLastError();
}

// Reduces the records, removing duplicates and storing the number of times
// each occurs in d_counts. The number of records after reduction is stored
// into host pointer num_records.
extern "C" int _system_poseidon2_deduplicate_records(
    Fp *d_records,
    uint32_t *d_counts,
    size_t *num_records
) {
    auto [grid, block] = kernel_launch_params(*num_records);
    FpArray<16> *d_records_fp16 = reinterpret_cast<FpArray<16> *>(d_records);
    size_t *d_num_records;

    // We want to sort and reduce the raw records, keeping track of how many
    // each occurs in d_counts. To prepare for reduce we need to a) allocate
    // d_num_records, b) fill d_counts with 1s, and c) group keys together
    // using sort.
    hipMallocAsync(&d_num_records, sizeof(size_t), hipStreamPerThread);
    hipMemcpyAsync(
        d_num_records, num_records, sizeof(size_t), hipMemcpyHostToDevice, hipStreamPerThread
    );
    fill_buffer<uint32_t><<<grid, block, 0, hipStreamPerThread>>>(d_counts, 1, *num_records);

    size_t sort_storage_bytes = 0;
    hipcub::DeviceMergeSort::SortKeys(
        nullptr,
        sort_storage_bytes,
        d_records_fp16,
        *num_records,
        Fp16CompareOp(),
        hipStreamPerThread
    );

    size_t reduce_storage_bytes = 0;
    hipcub::DeviceReduce::ReduceByKey(
        nullptr,
        reduce_storage_bytes,
        d_records_fp16,
        d_records_fp16,
        d_counts,
        d_counts,
        d_num_records,
        std::plus(),
        *num_records,
        hipStreamPerThread
    );

    size_t temp_storage_bytes = std::max(sort_storage_bytes, reduce_storage_bytes);
    void *d_temp_storage = nullptr;
    hipMallocAsync(&d_temp_storage, temp_storage_bytes, hipStreamPerThread);

    // TODO: We currently can't use DeviceRadixSort since each key is 64 bytes
    // which causes Fp16Decomposer usage to exceed shared memory. We need to
    // investigate better ways to sort, as merge sort is comparison-based.
    hipcub::DeviceMergeSort::SortKeys(
        d_temp_storage,
        temp_storage_bytes,
        d_records_fp16,
        *num_records,
        Fp16CompareOp(),
        hipStreamPerThread
    );

    // Removes duplicate values from d_records, and stores the number of times
    // they occur in d_counts. The number of unique values is stored into
    // d_num_records.
    hipcub::DeviceReduce::ReduceByKey(
        d_temp_storage,
        temp_storage_bytes,
        d_records_fp16,
        d_records_fp16,
        d_counts,
        d_counts,
        d_num_records,
        std::plus(),
        *num_records,
        hipStreamPerThread
    );

    hipMemcpyAsync(
        num_records, d_num_records, sizeof(size_t), hipMemcpyDeviceToHost, hipStreamPerThread
    );
    hipFreeAsync(d_num_records, hipStreamPerThread);
    hipFreeAsync(d_temp_storage, hipStreamPerThread);
    return hipGetLastError();
}
